#include "hip/hip_runtime.h"
// Copied from ggml-cuda/convert.cu and other llama.cpp files (as of 2024-05-20,
// revision 917dc8cf).  Used under the terms of LICENSE_GGML.

// PyTorch adds `-D` flags to disable fp16 conversions and operators when
// building extensions.  We want those conversions to be present because the
// copied GGML code uses them.  We don't include any PyTorch headers here.
#ifdef __CUDA_NO_HALF_CONVERSIONS__
# undef __CUDA_NO_HALF_CONVERSIONS__
#endif
//#ifdef __CUDA_NO_HALF_OPERATORS__
//# undef __CUDA_NO_HALF_OPERATORS__
//#endif
//#ifdef __CUDA_NO_HALF2_OPERATORS__
//# undef __CUDA_NO_HALF2_OPERATORS__
//#endif

#include "quant_formats.h"

#include <hip/hip_fp16.h>


//================================== k-quants

template<typename dst_t>
static __global__ void dequantize_block_q2_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const int64_t i   = blockIdx.x;
    const block_q2_K * x = (const block_q2_K *) vx;

    const int64_t tid = threadIdx.x;
#if QK_K == 256
    const int64_t n   = tid/32;
    const int64_t l   = tid - 32*n;
    const int64_t is  = 8*n + l/16;

    const uint8_t q = x[i].qs[32*n + l];
    dst_t * y = yy + i*QK_K + 128*n;

    float dall = __low2half(x[i].dm);
    float dmin = __high2half(x[i].dm);
    y[l+ 0] = dall * (x[i].scales[is+0] & 0xF) * ((q >> 0) & 3) - dmin * (x[i].scales[is+0] >> 4);
    y[l+32] = dall * (x[i].scales[is+2] & 0xF) * ((q >> 2) & 3) - dmin * (x[i].scales[is+2] >> 4);
    y[l+64] = dall * (x[i].scales[is+4] & 0xF) * ((q >> 4) & 3) - dmin * (x[i].scales[is+4] >> 4);
    y[l+96] = dall * (x[i].scales[is+6] & 0xF) * ((q >> 6) & 3) - dmin * (x[i].scales[is+6] >> 4);
#else
    const int64_t is = tid/16;  // 0 or 1
    const int64_t il = tid%16;  // 0...15
    const uint8_t q = x[i].qs[il] >> (2*is);
    dst_t * y = yy + i*QK_K + 16*is + il;
    float dall = __low2half(x[i].dm);
    float dmin = __high2half(x[i].dm);
    y[ 0] = dall * (x[i].scales[is+0] & 0xF) * ((q >> 0) & 3) - dmin * (x[i].scales[is+0] >> 4);
    y[32] = dall * (x[i].scales[is+2] & 0xF) * ((q >> 4) & 3) - dmin * (x[i].scales[is+2] >> 4);
#endif

}

template<typename dst_t>
static __global__ void dequantize_block_q3_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const int64_t i = blockIdx.x;
    const block_q3_K * x = (const block_q3_K *) vx;

#if QK_K == 256
    const int64_t r = threadIdx.x/4;
    const int64_t tid = r/2;
    const int64_t is0 = r%2;
    const int64_t l0 = 16*is0 + 4*(threadIdx.x%4);
    const int64_t n = tid / 4;
    const int64_t j = tid - 4*n;

    uint8_t m = 1 << (4*n + j);
    int64_t is = 8*n + 2*j + is0;
    int shift = 2*j;

    int8_t us = is <  4 ? (x[i].scales[is-0] & 0xF) | (((x[i].scales[is+8] >> 0) & 3) << 4) :
                is <  8 ? (x[i].scales[is-0] & 0xF) | (((x[i].scales[is+4] >> 2) & 3) << 4) :
                is < 12 ? (x[i].scales[is-8] >>  4) | (((x[i].scales[is+0] >> 4) & 3) << 4) :
                          (x[i].scales[is-8] >>  4) | (((x[i].scales[is-4] >> 6) & 3) << 4);
    float d_all = x[i].d;
    float dl = d_all * (us - 32);

    dst_t * y = yy + i*QK_K + 128*n + 32*j;
    const uint8_t * q = x[i].qs + 32*n;
    const uint8_t * hm = x[i].hmask;

    for (int l = l0; l < l0+4; ++l) y[l] = dl * ((int8_t)((q[l] >> shift) & 3) - ((hm[l] & m) ? 0 : 4));
#else
    const int64_t tid = threadIdx.x;
    const int64_t is  = tid/16;  // 0 or 1
    const int64_t il  = tid%16;  // 0...15
    const int64_t im  = il/8;    // 0...1
    const int64_t in  = il%8;    // 0...7

    dst_t * y = yy + i*QK_K + 16*is + il;

    const uint8_t q = x[i].qs[il] >> (2*is);
    const uint8_t h = x[i].hmask[in] >> (2*is + im);
    const float   d = (float)x[i].d;

    if (is == 0) {
        y[ 0] = d * ((x[i].scales[0] & 0xF) - 8) * ((int8_t)((q >> 0) & 3) - ((h >> 0) & 1 ? 0 : 4));
        y[32] = d * ((x[i].scales[1] & 0xF) - 8) * ((int8_t)((q >> 4) & 3) - ((h >> 4) & 1 ? 0 : 4));
    } else {
        y[ 0] = d * ((x[i].scales[0] >>  4) - 8) * ((int8_t)((q >> 0) & 3) - ((h >> 0) & 1 ? 0 : 4));
        y[32] = d * ((x[i].scales[1] >>  4) - 8) * ((int8_t)((q >> 4) & 3) - ((h >> 4) & 1 ? 0 : 4));
    }
#endif

}

#if QK_K == 256
static inline __device__ void get_scale_min_k4(int j, const uint8_t * q, uint8_t & d, uint8_t & m) {
    if (j < 4) {
        d = q[j] & 63; m = q[j + 4] & 63;
    } else {
        d = (q[j+4] & 0xF) | ((q[j-4] >> 6) << 4);
        m = (q[j+4] >>  4) | ((q[j-0] >> 6) << 4);
    }
}
#endif

template<typename dst_t>
static __global__ void dequantize_block_q4_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q4_K * x = (const block_q4_K *) vx;

    const int64_t i = blockIdx.x;

#if QK_K == 256
    // assume 32 threads
    const int64_t tid = threadIdx.x;
    const int64_t il  = tid/8;
    const int64_t ir  = tid%8;
    const int64_t is  = 2*il;
    const int64_t n   = 4;

    dst_t * y = yy + i*QK_K + 64*il + n*ir;

    const float dall = __low2half(x[i].dm);
    const float dmin = __high2half(x[i].dm);

    const uint8_t * q = x[i].qs + 32*il + n*ir;

    uint8_t sc, m;
    get_scale_min_k4(is + 0, x[i].scales, sc, m);
    const float d1 = dall * sc; const float m1 = dmin * m;
    get_scale_min_k4(is + 1, x[i].scales, sc, m);
    const float d2 = dall * sc; const float m2 = dmin * m;
    for (int l = 0; l < n; ++l) {
        y[l + 0] = d1 * (q[l] & 0xF) - m1;
        y[l +32] = d2 * (q[l] >>  4) - m2;
    }
#else
    const int64_t tid = threadIdx.x;
    const uint8_t * q = x[i].qs;
    dst_t * y = yy + i*QK_K;
    const float d = (float)x[i].dm[0];
    const float m = (float)x[i].dm[1];
    y[tid+ 0] = d * (x[i].scales[0] & 0xF) * (q[tid] & 0xF) - m * (x[i].scales[0] >> 4);
    y[tid+32] = d * (x[i].scales[1] & 0xF) * (q[tid] >>  4) - m * (x[i].scales[1] >> 4);
#endif
}

template<typename dst_t>
static __global__ void dequantize_block_q5_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q5_K * x = (const block_q5_K *) vx;

    const int64_t i = blockIdx.x;

#if QK_K == 256
    // assume 64 threads - this is very slightly better than the one below
    const int64_t tid = threadIdx.x;
    const int64_t il  = tid/16;   // il is in 0...3
    const int64_t ir  = tid%16;   // ir is in 0...15
    const int64_t is  = 2*il;     // is is in 0...6

    dst_t * y = yy + i*QK_K + 64*il + 2*ir;

    const float dall = __low2half(x[i].dm);
    const float dmin = __high2half(x[i].dm);

    const uint8_t * ql = x[i].qs + 32*il + 2*ir;
    const uint8_t * qh = x[i].qh + 2*ir;

    uint8_t sc, m;
    get_scale_min_k4(is + 0, x[i].scales, sc, m);
    const float d1 = dall * sc; const float m1 = dmin * m;
    get_scale_min_k4(is + 1, x[i].scales, sc, m);
    const float d2 = dall * sc; const float m2 = dmin * m;

    uint8_t   hm  = 1 << (2*il);
    y[ 0] = d1 * ((ql[ 0] & 0xF) + (qh[ 0] & hm ? 16 : 0)) - m1;
    y[ 1] = d1 * ((ql[ 1] & 0xF) + (qh[ 1] & hm ? 16 : 0)) - m1;
    hm <<= 1;
    y[32] = d2 * ((ql[ 0] >>  4) + (qh[ 0] & hm ? 16 : 0)) - m2;
    y[33] = d2 * ((ql[ 1] >>  4) + (qh[ 1] & hm ? 16 : 0)) - m2;
#else
    const int64_t tid = threadIdx.x;
    const uint8_t q = x[i].qs[tid];
    const int64_t im = tid/8;  // 0...3
    const int64_t in = tid%8;  // 0...7
    const int64_t is = tid/16; // 0 or 1
    const uint8_t h = x[i].qh[in] >> im;
    const float d = x[i].d;
    dst_t * y = yy + i*QK_K + tid;
    y[ 0] = d * x[i].scales[is+0] * ((q & 0xF) - ((h >> 0) & 1 ? 0 : 16));
    y[32] = d * x[i].scales[is+2] * ((q >>  4) - ((h >> 4) & 1 ? 0 : 16));
#endif
}

template<typename dst_t>
static __global__ void dequantize_block_q6_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q6_K * x = (const block_q6_K *) vx;

    const int64_t i = blockIdx.x;
#if QK_K == 256

    // assume 64 threads - this is very slightly better than the one below
    const int64_t tid = threadIdx.x;
    const int64_t ip  = tid/32;   // ip is 0 or 1
    const int64_t il  = tid - 32*ip; // 0...32
    const int64_t is  = 8*ip + il/16;

    dst_t * y = yy + i*QK_K + 128*ip + il;

    const float d = x[i].d;

    const uint8_t * ql = x[i].ql + 64*ip + il;
    const uint8_t   qh = x[i].qh[32*ip + il];
    const int8_t  * sc = x[i].scales + is;

    y[ 0] = d * sc[0] * ((int8_t)((ql[ 0] & 0xF) | (((qh >> 0) & 3) << 4)) - 32);
    y[32] = d * sc[2] * ((int8_t)((ql[32] & 0xF) | (((qh >> 2) & 3) << 4)) - 32);
    y[64] = d * sc[4] * ((int8_t)((ql[ 0]  >> 4) | (((qh >> 4) & 3) << 4)) - 32);
    y[96] = d * sc[6] * ((int8_t)((ql[32]  >> 4) | (((qh >> 6) & 3) << 4)) - 32);
#else

    // assume 32 threads
    const int64_t tid = threadIdx.x;
    const int64_t ip  = tid/16;         // 0 or 1
    const int64_t il  = tid - 16*ip;    // 0...15

    dst_t * y = yy + i*QK_K + 16*ip + il;

    const float d = x[i].d;

    const uint8_t   ql = x[i].ql[16*ip + il];
    const uint8_t   qh = x[i].qh[il] >> (2*ip);
    const int8_t  * sc = x[i].scales;

    y[ 0] = d * sc[ip+0] * ((int8_t)((ql & 0xF) | (((qh >> 0) & 3) << 4)) - 32);
    y[32] = d * sc[ip+2] * ((int8_t)((ql  >> 4) | (((qh >> 4) & 3) << 4)) - 32);
#endif
}


template<typename dst_t>
static void dequantize_row_q2_K_cuda(const void * vx, dst_t * y, const int64_t k, hipStream_t stream) {
    const int nb = k / QK_K;
#if QK_K == 256
    dequantize_block_q2_K<<<nb, 64, 0, stream>>>(vx, y);
#else
    dequantize_block_q2_K<<<nb, 32, 0, stream>>>(vx, y);
#endif
}

template<typename dst_t>
static void dequantize_row_q3_K_cuda(const void * vx, dst_t * y, const int64_t k, hipStream_t stream) {
    const int nb = k / QK_K;
#if QK_K == 256
    dequantize_block_q3_K<<<nb, 64, 0, stream>>>(vx, y);
#else
    dequantize_block_q3_K<<<nb, 32, 0, stream>>>(vx, y);
#endif
}

template<typename dst_t>
static void dequantize_row_q4_0_cuda(const void * vx, dst_t * y, const int64_t k, hipStream_t stream) {
    const int nb32 = k / 32;
    const int nb = (k + 255) / 256;
    dequantize_block_q4_0<<<nb, 32, 0, stream>>>(vx, y, nb32);
}

template<typename dst_t>
static void dequantize_row_q4_1_cuda(const void * vx, dst_t * y, const int64_t k, hipStream_t stream) {
    const int nb32 = k / 32;
    const int nb = (k + 255) / 256;
    dequantize_block_q4_1<<<nb, 32, 0, stream>>>(vx, y, nb32);
}

template<typename dst_t>
static void dequantize_row_q4_K_cuda(const void * vx, dst_t * y, const int64_t k, hipStream_t stream) {
    const int nb = k / QK_K;
    dequantize_block_q4_K<<<nb, 32, 0, stream>>>(vx, y);
}

template<typename dst_t>
static void dequantize_row_q5_K_cuda(const void * vx, dst_t * y, const int64_t k, hipStream_t stream) {
    const int nb = k / QK_K;
#if QK_K == 256
    dequantize_block_q5_K<<<nb, 64, 0, stream>>>(vx, y);
#else
    dequantize_block_q5_K<<<nb, 32, 0, stream>>>(vx, y);
#endif
}

template<typename dst_t>
static void dequantize_row_q6_K_cuda(const void * vx, dst_t * y, const int64_t k, hipStream_t stream) {
    const int nb = k / QK_K;
#if QK_K == 256
    dequantize_block_q6_K<<<nb, 64, 0, stream>>>(vx, y);
#else
    dequantize_block_q6_K<<<nb, 32, 0, stream>>>(vx, y);
#endif
}


void dequantize_row_q2_K_cuda_fp16(
        const void * vx, ggml_half_placeholder * y, const int64_t k, hipStream_t stream) {
    dequantize_row_q2_K_cuda(vx, (ggml_half*)y, k, stream);
}

void dequantize_row_q3_K_cuda_fp16(
        const void * vx, ggml_half_placeholder * y, const int64_t k, hipStream_t stream) {
    dequantize_row_q3_K_cuda(vx, (ggml_half*)y, k, stream);
}

void dequantize_row_q4_K_cuda_fp16(
        const void * vx, ggml_half_placeholder * y, const int64_t k, hipStream_t stream) {
    dequantize_row_q4_K_cuda(vx, (ggml_half*)y, k, stream);
}

void dequantize_row_q5_K_cuda_fp16(
        const void * vx, ggml_half_placeholder * y, const int64_t k, hipStream_t stream) {
    dequantize_row_q5_K_cuda(vx, (ggml_half*)y, k, stream);
}

void dequantize_row_q6_K_cuda_fp16(
        const void * vx, ggml_half_placeholder * y, const int64_t k, hipStream_t stream) {
    dequantize_row_q6_K_cuda(vx, (ggml_half*)y, k, stream);
}


void dequantize_row_q2_K_cuda_fp32(const void * vx, float * y, const int64_t k, hipStream_t stream) {
    dequantize_row_q2_K_cuda(vx, y, k, stream);
}

void dequantize_row_q3_K_cuda_fp32(const void * vx, float * y, const int64_t k, hipStream_t stream) {
    dequantize_row_q3_K_cuda(vx, y, k, stream);
}

void dequantize_row_q4_K_cuda_fp32(const void * vx, float * y, const int64_t k, hipStream_t stream) {
    dequantize_row_q4_K_cuda(vx, y, k, stream);
}

void dequantize_row_q5_K_cuda_fp32(const void * vx, float * y, const int64_t k, hipStream_t stream) {
    dequantize_row_q5_K_cuda(vx, y, k, stream);
}

void dequantize_row_q6_K_cuda_fp32(const void * vx, float * y, const int64_t k, hipStream_t stream) {
    dequantize_row_q6_K_cuda(vx, y, k, stream);
}
